/* 
Author: Liming Zhao (zlmzju@gmail.com)
*/
#include "./reorder-inl.h"
namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(ReorderParam param, int dtype) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
	  op = new ReorderOp<gpu, DType>(param);
  })
  return op;
}
}  // namespace op
}  // namespace mxnet